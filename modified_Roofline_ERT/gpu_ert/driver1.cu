#include <hip/hip_runtime.h>
#include <inttypes.h>
#include <iostream>
#include <stdint.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>

#include <sys/shm.h>
#include <unistd.h>

int flop = 0;
// helper functions and utilities to work with CUDA
#define ERT_FLOP 2
#define ERT_TRIALS_MIN 1
#define ERT_WORKING_SET_MIN 1
#define GBUNIT (1024 * 1024 * 1024)

#define REP2(S)                                                                \
    S;                                                                         \
    S
#define REP4(S)                                                                \
    REP2(S);                                                                   \
    REP2(S)
#define REP8(S)                                                                \
    REP4(S);                                                                   \
    REP4(S)
#define REP16(S)                                                               \
    REP8(S);                                                                   \
    REP8(S)
#define REP32(S)                                                               \
    REP16(S);                                                                  \
    REP16(S)
#define REP64(S)                                                               \
    REP32(S);                                                                  \
    REP32(S)
#define REP128(S)                                                              \
    REP64(S);                                                                  \
    REP64(S)
#define REP256(S)                                                              \
    REP128(S);                                                                 \
    REP128(S)
#define REP512(S)                                                              \
    REP256(S);                                                                 \
    REP256(S)

#define KERNEL2(a, b, c) ((a) = (a) * (b) + (c))
#define KERNEL1(a, b, c) ((a) = (b) + (c))

int gpu_blocks = 512;
int gpu_threads = 512;

#ifdef BW_PCT

#if BW_PCT == 100
int flop = 2;
#endif
#if BW_PCT == 90
int flop = 4;
#endif

#if BW_PCT == 80
int flop = 6;
#endif

#if BW_PCT == 70
int flop = 8;
#endif

#if BW_PCT == 60
int flop = 8;

#endif

#if BW_PCT == 50
int flop = 10;

#endif

#if BW_PCT == 40
int flop = 12;
#endif

#if BW_PCT == 30
int flop = 16;
#endif

#if BW_PCT == 20
int flop = 24;
#endif

#if BW_PCT == 10
int flop = 52;
#endif

#endif

#ifdef INTENS

#if INTENS == 1
int flop = 1;
#endif
#if INTENS == 2
int flop = 2;
#endif

#if INTENS == 3
int flop = 4;
#endif

#if INTENS == 4
int flop = 6;
#endif

#if INTENS == 5
int flop = 8;

#endif

#if INTENS == 6
int flop = 12;

#endif

#if INTENS == 7
int flop = 16;
#endif

#if INTENS == 8
int flop = 20;
#endif

#if INTENS == 9
int flop = 28;
#endif

#if INTENS == 10
int flop = 32;
#endif

#if INTENS == 11
int flop = 48;
#endif

#if INTENS == 12
int flop = 64;
#endif

#if INTENS == 13
int flop = 96;
#endif

#if INTENS == 14
int flop = 128;
#endif

#if INTENS == 15
int flop = 192;

#endif

#if INTENS == 16
int flop = 256;

#endif

#if INTENS == 17
int flop = 384;
#endif

#if INTENS == 18
int flop = 512;
#endif

#if INTENS == 19
int flop = 768;
#endif

#if INTENS == 20
int flop = 1024;
#endif

#endif

void initialize(uint64_t nsize, double *__restrict__ A, double value) {
    uint64_t i;
    for (i = 0; i < nsize; ++i) {
        A[i] = value;
    }
}

void gpuKernel(uint64_t nsize, uint64_t ntrials, double *__restrict__ array,
               int *bytes_per_elem, int *mem_accesses_per_elem);

__global__ void block_stride(uint64_t ntrials, uint64_t nsize, double *A) {
    uint64_t total_thr = gridDim.x * blockDim.x;
    uint64_t elem_per_thr = (nsize + (total_thr - 1)) / total_thr;
    uint64_t blockOffset = blockIdx.x * blockDim.x;

    uint64_t start_idx = blockOffset + threadIdx.x;
    uint64_t end_idx = start_idx + elem_per_thr * total_thr;
    uint64_t stride_idx = total_thr;

    if (start_idx > nsize) {
        start_idx = nsize;
    }

    if (end_idx > nsize) {
        end_idx = nsize;
    }

    double alpha = 0.5;
    uint64_t i, j;
    for (j = 0; j < ntrials; ++j) {
        for (i = start_idx; i < end_idx; i += stride_idx) {
            double beta = 0.8;

#ifdef BW_PCT
#if BW_PCT == 100

            /* add 1 flop */
            // KERNEL1(beta,A[i],alpha);
            // KERNEL1(beta,A[i],alpha);
            /* add 2 flops */
            KERNEL2(beta, A[i], alpha);
            /* add 4 flops */
            // REP2(KERNEL2(beta,A[i],alpha));

            /* add 8 flops */
            // REP4(KERNEL2(beta,A[i],alpha));
            /* add 16 flops */
            // REP8(KERNEL2(beta,A[i],alpha));
            /* add 32 flops */
            // REP16(KERNEL2(beta,A[i],alpha));
            /* add 64 flops */
            // REP32(KERNEL2(beta,A[i],alpha));
            /* add 128 flops */
            // REP64(KERNEL2(beta,A[i],alpha));
            /* add 256 flops */
            // REP128(KERNEL2(beta,A[i],alpha));
            /* add 512 flops */
            // REP256(KERNEL2(beta,A[i],alpha));
            /* add 1024 flops */
            // REP512(KERNEL2(beta,A[i],alpha));

#endif
#if BW_PCT == 90
            /* add 1 flop */
            // KERNEL1(beta,A[i],alpha);
            // KERNEL1(beta,A[i],alpha);
            /* add 2 flops */
            // KERNEL2(beta,A[i],alpha);
            /* add 4 flops */
            REP2(KERNEL2(beta, A[i], alpha));

            /* add 8 flops */
            // REP4(KERNEL2(beta,A[i],alpha));
            /* add 16 flops */
            // REP8(KERNEL2(beta,A[i],alpha));
            /* add 32 flops */
            // REP16(KERNEL2(beta,A[i],alpha));
            /* add 64 flops */
            // REP32(KERNEL2(beta,A[i],alpha));
            /* add 128 flops */
            // REP64(KERNEL2(beta,A[i],alpha));
            /* add 256 flops */
            // REP128(KERNEL2(beta,A[i],alpha));
            /* add 512 flops */
            // REP256(KERNEL2(beta,A[i],alpha));
            /* add 1024 flops */
            // REP512(KERNEL2(beta,A[i],alpha));

#endif

#if BW_PCT == 80
            /* add 1 flop */
            // KERNEL1(beta,A[i],alpha);
            // KERNEL1(beta,A[i],alpha);
            /* add 2 flops */
            KERNEL2(beta, A[i], alpha);
            /* add 4 flops */
            REP2(KERNEL2(beta, A[i], alpha));

            /* add 8 flops */
            // REP4(KERNEL2(beta,A[i],alpha));
            /* add 16 flops */
            // REP8(KERNEL2(beta,A[i],alpha));
            /* add 32 flops */
            // REP16(KERNEL2(beta,A[i],alpha));
            /* add 64 flops */
            // REP32(KERNEL2(beta,A[i],alpha));
            /* add 128 flops */
            // REP64(KERNEL2(beta,A[i],alpha));
            /* add 256 flops */
            // REP128(KERNEL2(beta,A[i],alpha));
            /* add 512 flops */
            // REP256(KERNEL2(beta,A[i],alpha));
            /* add 1024 flops */
            // REP512(KERNEL2(beta,A[i],alpha));

#endif

#if BW_PCT == 70
            /* add 1 flop */
            // KERNEL1(beta,A[i],alpha);
            // KERNEL1(beta,A[i],alpha);
            /* add 2 flops */
            // KERNEL2(beta,A[i],alpha);
            /* add 4 flops */
            // REP2(KERNEL2(beta,A[i],alpha));
            // REP2(KERNEL2(beta,A[i],alpha));

            /* add 8 flops */
            REP4(KERNEL2(beta, A[i], alpha));
            /* add 16 flops */
            // REP8(KERNEL2(beta,A[i],alpha));
            /* add 32 flops */
            // REP16(KERNEL2(beta,A[i],alpha));
            /* add 64 flops */
            // REP32(KERNEL2(beta,A[i],alpha));
            /* add 128 flops */
            // REP64(KERNEL2(beta,A[i],alpha));
            /* add 256 flops */
            // REP128(KERNEL2(beta,A[i],alpha));
            /* add 512 flops */
            // REP256(KERNEL2(beta,A[i],alpha));
            /* add 1024 flops */
            // REP512(KERNEL2(beta,A[i],alpha));

#endif

#if BW_PCT == 60
            /* add 1 flop */
            // KERNEL1(beta,A[i],alpha);
            // KERNEL1(beta,A[i],alpha);
            /* add 2 flops */
            // KERNEL2(beta,A[i],alpha);
            // KERNEL2(beta,A[i],alpha);
            /* add 4 flops */
            // REP2(KERNEL2(beta,A[i],alpha));

            /* add 8 flops */
            REP4(KERNEL2(beta, A[i], alpha));
            /* add 16 flops */
            // REP8(KERNEL2(beta,A[i],alpha));
            /* add 32 flops */
            // REP16(KERNEL2(beta,A[i],alpha));
            /* add 64 flops */
            // REP32(KERNEL2(beta,A[i],alpha));
            /* add 128 flops */
            // REP64(KERNEL2(beta,A[i],alpha));
            /* add 256 flops */
            // REP128(KERNEL2(beta,A[i],alpha));
            /* add 512 flops */
            // REP256(KERNEL2(beta,A[i],alpha));
            /* add 1024 flops */
            // REP512(KERNEL2(beta,A[i],alpha));

#endif

#if BW_PCT == 50
            /* add 1 flop */
            // KERNEL1(beta,A[i],alpha);
            // KERNEL1(beta,A[i],alpha);
            /* add 2 flops */
            KERNEL2(beta, A[i], alpha);
            /* add 4 flops */
            // REP2(KERNEL2(beta,A[i],alpha));

            /* add 8 flops */
            REP4(KERNEL2(beta, A[i], alpha));
            /* add 16 flops */
            // REP8(KERNEL2(beta,A[i],alpha));
            /* add 32 flops */
            // REP16(KERNEL2(beta,A[i],alpha));
            /* add 64 flops */
            // REP32(KERNEL2(beta,A[i],alpha));
            /* add 128 flops */
            // REP64(KERNEL2(beta,A[i],alpha));
            /* add 256 flops */
            // REP128(KERNEL2(beta,A[i],alpha));
            /* add 512 flops */
            // REP256(KERNEL2(beta,A[i],alpha));
            /* add 1024 flops */
            // REP512(KERNEL2(beta,A[i],alpha));

#endif

#if BW_PCT == 40
            /* add 1 flop */
            // KERNEL1(beta,A[i],alpha);
            // KERNEL1(beta,A[i],alpha);
            /* add 2 flops */
            // KERNEL2(beta,A[i],alpha);
            /* add 4 flops */
            REP2(KERNEL2(beta, A[i], alpha));
            /* add 8 flops */
            REP4(KERNEL2(beta, A[i], alpha));
            /* add 16 flops */
            // REP8(KERNEL2(beta,A[i],alpha));
            /* add 32 flops */
            // REP16(KERNEL2(beta,A[i],alpha));
            /* add 64 flops */
            // REP32(KERNEL2(beta,A[i],alpha));
            /* add 128 flops */
            // REP64(KERNEL2(beta,A[i],alpha));
            /* add 256 flops */
            // REP128(KERNEL2(beta,A[i],alpha));
            /* add 512 flops */
            // REP256(KERNEL2(beta,A[i],alpha));
            /* add 1024 flops */
            // REP512(KERNEL2(beta,A[i],alpha));

#endif

#if BW_PCT == 30
            /* add 1 flop */
            // KERNEL1(beta,A[i],alpha);
            // KERNEL1(beta,A[i],alpha);
            /* add 2 flops */
            // KERNEL2(beta,A[i],alpha);
            /* add 4 flops */
            // REP2(KERNEL2(beta,A[i],alpha));

            /* add 8 flops */
            // REP4(KERNEL2(beta,A[i],alpha));
            /* add 16 flops */
            REP8(KERNEL2(beta, A[i], alpha));
            /* add 32 flops */
            // REP16(KERNEL2(beta,A[i],alpha));
            /* add 64 flops */
            // REP32(KERNEL2(beta,A[i],alpha));
            /* add 128 flops */
            // REP64(KERNEL2(beta,A[i],alpha));
            /* add 256 flops */
            // REP128(KERNEL2(beta,A[i],alpha));
            /* add 512 flops */
            // REP256(KERNEL2(beta,A[i],alpha));
            /* add 1024 flops */
            // REP512(KERNEL2(beta,A[i],alpha));

#endif

#if BW_PCT == 20
            /* add 1 flop */
            // KERNEL1(beta,A[i],alpha);
            // KERNEL1(beta,A[i],alpha);
            /* add 2 flops */
            // KERNEL2(beta,A[i],alpha);
            /* add 4 flops */
            // REP2(KERNEL2(beta,A[i],alpha));

            /* add 8 flops */
            REP4(KERNEL2(beta, A[i], alpha));
            /* add 16 flops */
            REP8(KERNEL2(beta, A[i], alpha));
            /* add 32 flops */
            // REP16(KERNEL2(beta,A[i],alpha));
            /* add 64 flops */
            // REP32(KERNEL2(beta,A[i],alpha));
            /* add 128 flops */
            // REP64(KERNEL2(beta,A[i],alpha));
            /* add 256 flops */
            // REP128(KERNEL2(beta,A[i],alpha));
            /* add 512 flops */
            // REP256(KERNEL2(beta,A[i],alpha));
            /* add 1024 flops */
            // REP512(KERNEL2(beta,A[i],alpha));

#endif

#if BW_PCT == 10
            /* add 1 flop */
            // KERNEL1(beta,A[i],alpha);
            // KERNEL1(beta,A[i],alpha);
            /* add 2 flops */
            // KERNEL2(beta,A[i],alpha);
            /* add 4 flops */
            REP2(KERNEL2(beta, A[i], alpha));

            /* add 8 flops */
            // REP4(KERNEL2(beta,A[i],alpha));
            /* add 16 flops */
            REP8(KERNEL2(beta, A[i], alpha));
            /* add 32 flops */
            REP16(KERNEL2(beta, A[i], alpha));
            /* add 64 flops */
            // REP32(KERNEL2(beta,A[i],alpha));
            /* add 128 flops */
            // REP64(KERNEL2(beta,A[i],alpha));
            /* add 256 flops */
            // REP128(KERNEL2(beta,A[i],alpha));
            /* add 512 flops */
            // REP256(KERNEL2(beta,A[i],alpha));
            /* add 1024 flops */
            // REP512(KERNEL2(beta,A[i],alpha));

#endif

#endif

#ifdef INTENS

#if INTENS == 1
            //#if (ERT_FLOP & 1) == 1       /* add 1 flop */
            KERNEL1(beta, A[i], alpha);

#endif

#if INTENS == 2
            //#if (ERT_FLOP & 2) == 2       /* add 2 flops */
            KERNEL2(beta, A[i], alpha);

#endif

#if INTENS == 3
            // int flop = 4;
            //#if (ERT_FLOP & 4) == 4       /* add 4 flops */
            REP2(KERNEL2(beta, A[i], alpha));

#endif

#if INTENS == 4
            // int flop = 6;
            //#if (ERT_FLOP & 2) == 2       /* add 2 flops */
            KERNEL2(beta, A[i], alpha);
            //#if (ERT_FLOP & 4) == 4       /* add 4 flops */
            REP2(KERNEL2(beta, A[i], alpha));

#endif

#if INTENS == 5
            // int flop = 8;
            //#if (ERT_FLOP & 8) == 8       /* add 8 flops */
            REP4(KERNEL2(beta, A[i], alpha));

#endif

#if INTENS == 6
            // int flop = 12;
            //#if (ERT_FLOP & 4) == 4       /* add 4 flops */
            REP2(KERNEL2(beta, A[i], alpha));
            //#if (ERT_FLOP & 8) == 8       /* add 8 flops */
            REP4(KERNEL2(beta, A[i], alpha));

#endif

#if INTENS == 7
            // int flop = 16;
            //#if (ERT_FLOP & 16) == 16     /* add 16 flops */
            REP8(KERNEL2(beta, A[i], alpha));

#endif

#if INTENS == 8
            // int flop = 20;

            //#if (ERT_FLOP & 4) == 4       /* add 4 flops */
            REP2(KERNEL2(beta, A[i], alpha));

            //#if (ERT_FLOP & 16) == 16     /* add 16 flops */
            REP8(KERNEL2(beta, A[i], alpha));

#endif

#if INTENS == 9
            // int flop = 28;

            //#if (ERT_FLOP & 4) == 4       /* add 4 flops */
            REP2(KERNEL2(beta, A[i], alpha));
            //#if (ERT_FLOP & 8) == 8       /* add 8 flops */
            REP4(KERNEL2(beta, A[i], alpha));
            //#if (ERT_FLOP & 16) == 16     /* add 16 flops */
            REP8(KERNEL2(beta, A[i], alpha));

#endif

#if INTENS == 10
            // int flop = 32;
            //#if (ERT_FLOP & 32) == 32     /* add 32 flops */
            REP16(KERNEL2(beta, A[i], alpha));

#endif

#if INTENS == 11
            // int flop = 48;
            //#if (ERT_FLOP & 16) == 16     /* add 16 flops */
            REP8(KERNEL2(beta, A[i], alpha));
            //#if (ERT_FLOP & 32) == 32     /* add 32 flops */
            REP16(KERNEL2(beta, A[i], alpha));

#endif

#if INTENS == 12
            // int flop = 64;
            //#if (ERT_FLOP & 64) == 64     /* add 64 flops */
            REP32(KERNEL2(beta, A[i], alpha));

#endif

#if INTENS == 13
            // int flop = 96;
            //#if (ERT_FLOP & 32) == 32     /* add 32 flops */
            REP16(KERNEL2(beta, A[i], alpha));
            //#if (ERT_FLOP & 64) == 64     /* add 64 flops */
            REP32(KERNEL2(beta, A[i], alpha));

#endif

#if INTENS == 14
            // int flop = 128;
            //#if (ERT_FLOP & 128) == 128   /* add 128 flops */
            REP64(KERNEL2(beta, A[i], alpha));
#endif

#if INTENS == 15
            // int flop = 192;
            //#if (ERT_FLOP & 64) == 64     /* add 64 flops */
            REP32(KERNEL2(beta, A[i], alpha));
            //#if (ERT_FLOP & 128) == 128   /* add 128 flops */
            REP64(KERNEL2(beta, A[i], alpha));

#endif

#if INTENS == 16
            // int flop = 256;
            //#if (ERT_FLOP & 256) == 256   /* add 256 flops */
            REP128(KERNEL2(beta, A[i], alpha));

#endif

#if INTENS == 17
            // int flop = 384;
            //#if (ERT_FLOP & 128) == 128   /* add 128 flops */
            REP64(KERNEL2(beta, A[i], alpha));
            //#if (ERT_FLOP & 256) == 256   /* add 256 flops */
            REP128(KERNEL2(beta, A[i], alpha));
#endif

#if INTENS == 18
            int flop = 512;
            //#if (ERT_FLOP & 512) == 512   /* add 512 flops */
            REP256(KERNEL2(beta, A[i], alpha));

#endif

#if INTENS == 19
            int flop = 768;
            //#if (ERT_FLOP & 256) == 256   /* add 256 flops */
            REP128(KERNEL2(beta, A[i], alpha));
            //#if (ERT_FLOP & 512) == 512   /* add 512 flops */
            REP256(KERNEL2(beta, A[i], alpha));

#endif

#if INTENS == 20
            int flop = 1024;
            //#if (ERT_FLOP & 1024) == 1024 /* add 1024 flops */
            REP512(KERNEL2(beta, A[i], alpha));

#endif

#endif

            /* add 1 flop */
            // KERNEL1(beta,A[i],alpha);
            // KERNEL1(beta,A[i],alpha);
            /* add 2 flops */
            // KERNEL2(beta,A[i],alpha);
            /* add 4 flops */
            // REP2(KERNEL2(beta,A[i],alpha));
            /* add 8 flops */
            // REP4(KERNEL2(beta,A[i],alpha));
            /* add 16 flops */
            // REP8(KERNEL2(beta,A[i],alpha));
            /* add 32 flops */
            // REP16(KERNEL2(beta,A[i],alpha));
            /* add 64 flops */
            // REP32(KERNEL2(beta,A[i],alpha));
            /* add 128 flops */
            // REP64(KERNEL2(beta,A[i],alpha));
            /* add 256 flops */
            // REP128(KERNEL2(beta,A[i],alpha));
            /* add 512 flops */
            // REP256(KERNEL2(beta,A[i],alpha));
            /* add 1024 flops */
            // REP512(KERNEL2(beta,A[i],alpha));

            A[i] = beta;
        }
        alpha = alpha * (1 - 1e-8);
    }
}

__global__ void block_stride5(uint64_t ntrials, uint64_t nsize, double *A) {
    uint64_t total_thr = gridDim.x * blockDim.x;
    uint64_t elem_per_thr = (nsize + (total_thr - 1)) / total_thr;
    uint64_t blockOffset = blockIdx.x * blockDim.x;

    uint64_t start_idx = blockOffset + threadIdx.x;
    uint64_t end_idx = start_idx + elem_per_thr * total_thr;
    uint64_t stride_idx = total_thr;

    if (start_idx > nsize) {
        start_idx = nsize;
    }

    if (end_idx > nsize) {
        end_idx = nsize;
    }

    double alpha = 0.5;
    uint64_t i, j;
    for (j = 0; j < ntrials; ++j) {
        for (i = start_idx; i < end_idx; i += stride_idx) {
            double beta = 0.8;

            //#if (ERT_FLOP & 16) == 16     /* add 16 flops */
            REP8(KERNEL2(beta, A[i], alpha));
            //#if (ERT_FLOP & 32) == 32     /* add 32 flops */
            REP16(KERNEL2(beta, A[i], alpha));

            A[i] = beta;
        }
        alpha = alpha * (1 - 1e-8);
    }
}

void gpuKernel5(uint64_t nsize, uint64_t ntrials, double *__restrict__ A,
                int *bytes_per_elem, int *mem_accesses_per_elem) {
    *bytes_per_elem = sizeof(*A);
    *mem_accesses_per_elem = 2;
    flop = 48;
    // gpu_blocks = (nsize+1023)/1024;
    block_stride5<<<gpu_blocks, gpu_threads>>>(ntrials, nsize, A);
}

__global__ void block_stride4(uint64_t ntrials, uint64_t nsize, double *A) {
    uint64_t total_thr = gridDim.x * blockDim.x;
    uint64_t elem_per_thr = (nsize + (total_thr - 1)) / total_thr;
    uint64_t blockOffset = blockIdx.x * blockDim.x;

    uint64_t start_idx = blockOffset + threadIdx.x;
    uint64_t end_idx = start_idx + elem_per_thr * total_thr;
    uint64_t stride_idx = total_thr;

    if (start_idx > nsize) {
        start_idx = nsize;
    }

    if (end_idx > nsize) {
        end_idx = nsize;
    }

    double alpha = 0.5;
    uint64_t i, j;
    for (j = 0; j < ntrials; ++j) {
        for (i = start_idx; i < end_idx; i += stride_idx) {
            double beta = 0.8;

            // int flop = 20;

            //#if (ERT_FLOP & 4) == 4       /* add 4 flops */
            REP2(KERNEL2(beta, A[i], alpha));

            //#if (ERT_FLOP & 16) == 16     /* add 16 flops */
            REP8(KERNEL2(beta, A[i], alpha));

            A[i] = beta;
        }
        alpha = alpha * (1 - 1e-8);
    }
}

void gpuKernel4(uint64_t nsize, uint64_t ntrials, double *__restrict__ A,
                int *bytes_per_elem, int *mem_accesses_per_elem) {
    *bytes_per_elem = sizeof(*A);
    *mem_accesses_per_elem = 2;
    flop = 20;
    // gpu_blocks = (nsize+1023)/1024;
    block_stride4<<<gpu_blocks, gpu_threads>>>(ntrials, nsize, A);
}

__global__ void block_stride3(uint64_t ntrials, uint64_t nsize, double *A) {
    uint64_t total_thr = gridDim.x * blockDim.x;
    uint64_t elem_per_thr = (nsize + (total_thr - 1)) / total_thr;
    uint64_t blockOffset = blockIdx.x * blockDim.x;

    uint64_t start_idx = blockOffset + threadIdx.x;
    uint64_t end_idx = start_idx + elem_per_thr * total_thr;
    uint64_t stride_idx = total_thr;

    if (start_idx > nsize) {
        start_idx = nsize;
    }

    if (end_idx > nsize) {
        end_idx = nsize;
    }

    double alpha = 0.5;
    uint64_t i, j;
    for (j = 0; j < ntrials; ++j) {
        for (i = start_idx; i < end_idx; i += stride_idx) {
            double beta = 0.8;

            // int flop = 12;
            //#if (ERT_FLOP & 4) == 4       /* add 4 flops */
            REP2(KERNEL2(beta, A[i], alpha));
            //#if (ERT_FLOP & 8) == 8       /* add 8 flops */
            REP4(KERNEL2(beta, A[i], alpha));

            A[i] = beta;
        }
        alpha = alpha * (1 - 1e-8);
    }
}

void gpuKernel3(uint64_t nsize, uint64_t ntrials, double *__restrict__ A,
                int *bytes_per_elem, int *mem_accesses_per_elem) {
    *bytes_per_elem = sizeof(*A);
    *mem_accesses_per_elem = 2;
    flop = 12;
    // gpu_blocks = (nsize+1023)/1024;
    block_stride3<<<gpu_blocks, gpu_threads>>>(ntrials, nsize, A);
}

__global__ void block_stride2(uint64_t ntrials, uint64_t nsize, double *A) {
    uint64_t total_thr = gridDim.x * blockDim.x;
    uint64_t elem_per_thr = (nsize + (total_thr - 1)) / total_thr;
    uint64_t blockOffset = blockIdx.x * blockDim.x;

    uint64_t start_idx = blockOffset + threadIdx.x;
    uint64_t end_idx = start_idx + elem_per_thr * total_thr;
    uint64_t stride_idx = total_thr;

    if (start_idx > nsize) {
        start_idx = nsize;
    }

    if (end_idx > nsize) {
        end_idx = nsize;
    }

    double alpha = 0.5;
    uint64_t i, j;
    for (j = 0; j < ntrials; ++j) {
        for (i = start_idx; i < end_idx; i += stride_idx) {
            double beta = 0.8;

            // int flop = 6;
            //#if (ERT_FLOP & 2) == 2       /* add 2 flops */
            KERNEL2(beta, A[i], alpha);
            //#if (ERT_FLOP & 4) == 4       /* add 4 flops */
            REP2(KERNEL2(beta, A[i], alpha));

            A[i] = beta;
        }
        alpha = alpha * (1 - 1e-8);
    }
}

void gpuKernel2(uint64_t nsize, uint64_t ntrials, double *__restrict__ A,
                int *bytes_per_elem, int *mem_accesses_per_elem) {
    *bytes_per_elem = sizeof(*A);
    *mem_accesses_per_elem = 2;
    flop = 6;
    // gpu_blocks = (nsize+1023)/1024;
    block_stride2<<<gpu_blocks, gpu_threads>>>(ntrials, nsize, A);
}

__global__ void block_stride1(uint64_t ntrials, uint64_t nsize, double *A) {
    uint64_t total_thr = gridDim.x * blockDim.x;
    uint64_t elem_per_thr = (nsize + (total_thr - 1)) / total_thr;
    uint64_t blockOffset = blockIdx.x * blockDim.x;

    uint64_t start_idx = blockOffset + threadIdx.x;
    uint64_t end_idx = start_idx + elem_per_thr * total_thr;
    uint64_t stride_idx = total_thr;

    if (start_idx > nsize) {
        start_idx = nsize;
    }

    if (end_idx > nsize) {
        end_idx = nsize;
    }

    double alpha = 0.5;
    uint64_t i, j;
    for (j = 0; j < ntrials; ++j) {
        for (i = start_idx; i < end_idx; i += stride_idx) {
            double beta = 0.8;

            //#if (ERT_FLOP & 1) == 1       /* add 1 flop */
            KERNEL1(beta, A[i], alpha);

            A[i] = beta;
        }
        alpha = alpha * (1 - 1e-8);
    }
}

void gpuKernel1(uint64_t nsize, uint64_t ntrials, double *__restrict__ A,
                int *bytes_per_elem, int *mem_accesses_per_elem) {
    *bytes_per_elem = sizeof(*A);
    *mem_accesses_per_elem = 2;
    flop = 1;
    // gpu_blocks = (nsize+1023)/1024;
    block_stride1<<<gpu_blocks, gpu_threads>>>(ntrials, nsize, A);
}

__global__ void block_stride_triad(uint64_t ntrials, uint64_t nsize, double *A) {
    uint64_t total_thr = gridDim.x * blockDim.x;
    uint64_t elem_per_thr = (nsize + (total_thr - 1)) / total_thr;
    uint64_t blockOffset = blockIdx.x * blockDim.x;

    uint64_t start_idx = blockOffset + threadIdx.x;
    uint64_t end_idx = start_idx + elem_per_thr * total_thr;
    uint64_t stride_idx = total_thr;

    if (start_idx > nsize) {
        start_idx = nsize;
    }

    if (end_idx > nsize) {
        end_idx = nsize;
    }

    double alpha = 0.5;
    uint64_t i, j;
    for (j = 0; j < ntrials; ++j) {
        for (i = start_idx; i < end_idx; i += stride_idx) {
            double beta = 0.8;

	    //#if (ERT_FLOP & 2) == 2       /* add 2 flops */
            KERNEL2(beta, A[i], alpha);

            A[i] = beta;
        }
        alpha = alpha * (1 - 1e-8);
    }
}

void gpuKernel_triad(uint64_t nsize, uint64_t ntrials, double *__restrict__ A,
                int *bytes_per_elem, int *mem_accesses_per_elem) {
    *bytes_per_elem = sizeof(*A);
    *mem_accesses_per_elem = 2;
    flop = 2;
    // gpu_blocks = (nsize+1023)/1024;
    block_stride_triad<<<gpu_blocks, gpu_threads>>>(ntrials, nsize, A);
}

void gpuKernel(uint64_t nsize, uint64_t ntrials, double *__restrict__ A,
               int *bytes_per_elem, int *mem_accesses_per_elem) {
    *bytes_per_elem = sizeof(*A);
    *mem_accesses_per_elem = 2;
    // gpu_blocks = (nsize+1023)/1024;
    block_stride<<<gpu_blocks, gpu_threads>>>(ntrials, nsize, A);
}

double getTime() {
    double time;
    struct timeval tm;
    gettimeofday(&tm, NULL);
    time = tm.tv_sec + (tm.tv_usec / 1000000.0);
    return time;
}

int main(int argc, char *argv[]) {

    key_t shmkey, shmkey1; /*      shared memory key       */
    int shmid, shmid1;     /*      shared memory id        */

    int *p_gpu, *p_cpu; /*      shared variable         */ /*shared */
    // shmdt (p);shmctl (shmid, IPC_RMID, 0);exit(0);

    /* initialize a shared variable in shared memory */
    // shmkey = ftok ("/home/66m", 5);       /* valid directory name and a
    // number */
    shmkey = ftok("/dev/null", 5);  /* valid directory name and a number */
    shmkey1 = ftok("/dev/null", 6); /* valid directory name and a number */
    // printf ("shmkey for p = %d\n", shmkey);

    shmid = shmget(shmkey, sizeof(int), 0644 | IPC_CREAT);
    shmid1 = shmget(shmkey1, sizeof(int), 0644 | IPC_CREAT);
    if (shmid < 0) { /* shared memory error check */
        perror("shmget\n");
        exit(1);
    }
    // pthread_mutex_t *m;
    // Mutex a(m,false);
    p_gpu = (int *)shmat(shmid, NULL, 0);  /* attach p to shared memory */
    p_cpu = (int *)shmat(shmid1, NULL, 0); /* attach p to shared memory */
    //printf("p_cpu=%d  p_gpu=%d is allocated in shared memory.\n\n", *p_cpu,*p_gpu);
    *p_gpu = 20;                            //*p = 0;
    if (*p_cpu != 20 && *p_cpu != 1) *p_cpu = -5;                           //*p = 0;

    //if (*p_cpu != 20) *p_cpu = -5;                           //*p = 0;
    //printf("p_cpu=%d  p_gpu=%d is allocated in shared memory.\n\n", *p_cpu,
           //*p_gpu);

    int rank = 0;
    int nprocs = 1;
    int nthreads = 1;
    int id = 0;

    uint64_t TSIZE = 1 << 30;
    uint64_t PSIZE = TSIZE / nprocs;

    double *buf = (double *)malloc(PSIZE);

    if (buf == NULL) {
        fprintf(stderr, "Out of memory!\n");
        return -1;
    }

    {
        id = 0;
        nthreads = 1;

        int num_gpus = 0;
        int gpu;
        int gpu_id;
        int numSMs;

        hipGetDeviceCount(&num_gpus);
        if (num_gpus < 1) {
            fprintf(stderr, "No CUDA device detected.\n");
            return -1;
        }

        for (gpu = 0; gpu < num_gpus; gpu++) {
            hipDeviceProp_t dprop;
            hipGetDeviceProperties(&dprop, gpu);
            /* printf("%d: %s\n",gpu,dprop.name); */
        }

        hipSetDevice(id % num_gpus);
        hipGetDevice(&gpu_id);
        hipDeviceGetAttribute(&numSMs, hipDeviceAttributeMultiprocessorCount, gpu_id);

        void (*fun_ptr)(uint64_t, uint64_t, double *, int *, int *);
        double *d_buf;
        for (int ar = 2; ar <= argc; ar++) {

            sleep(5);

            // if (id == 0) printf("Current kernel:  %s\n", argv[ar-1]);
            if (strcmp(argv[ar - 1], "1") == 0)
                fun_ptr = &gpuKernel1;
            if (strcmp(argv[ar - 1], "2") == 0)
                fun_ptr = &gpuKernel2;
            if (strcmp(argv[ar - 1], "3") == 0)
                fun_ptr = &gpuKernel3;
            if (strcmp(argv[ar - 1], "4") == 0)
                fun_ptr = &gpuKernel4;
            if (strcmp(argv[ar - 1], "5") == 0)
                fun_ptr = &gpuKernel5;
            if (strcmp(argv[ar - 1], "6") == 0)
                fun_ptr = &gpuKernel_triad;


            uint64_t nsize = PSIZE / nthreads;
            nsize = nsize & (~(64 - 1));
            nsize = nsize / sizeof(double);
            uint64_t nid = nsize * id;

            // initialize small chunck of buffer within each thread
            initialize(nsize, &buf[nid], 1.0);

            hipMalloc((void **)&d_buf, nsize * sizeof(double));
            hipMemset(d_buf, 0, nsize * sizeof(double));
            hipDeviceSynchronize();

            double startTime, endTime;
            uint64_t n;
            uint64_t t;
            int bytes_per_elem;
            int mem_accesses_per_elem;
            double seconds = 0;
            uint64_t working_set_size = 0;
            uint64_t total_bytes = 0;
            uint64_t total_flops = 0;

            n = 1 << 22;
            n = n * 8;

            // int lim = 35;
            int lim;
#ifdef LIM
            lim = LIM;
#endif
            while (n <= nsize) { // working set - nsize

                uint64_t ntrials = nsize / n;
                if (ntrials < 1)
                    ntrials = 1;

                // warmup begins
                /*
                cudaMemcpy(d_buf, &buf[nid], n*sizeof(double),
                cudaMemcpyHostToDevice); cudaDeviceSynchronize(); for (t = 1; t
                <= 30; t = t + 1) { gpuKernel(n, t, d_buf, &bytes_per_elem,
                &mem_accesses_per_elem); cudaDeviceSynchronize();

                }
                cudaMemcpy(&buf[nid], d_buf, n*sizeof(double),
                cudaMemcpyDeviceToHost); cudaDeviceSynchronize();
                */
                // warmup ends

                for (t = lim - 35; t <= lim;
                     t = t + 1) { // working set - ntrials
                    hipMemcpy(d_buf, &buf[nid], n * sizeof(double),
                               hipMemcpyHostToDevice);
                    hipDeviceSynchronize();

                    if (t >= (lim - 19)) {
                        *p_gpu = 10;
                        while (1) {
                            if (*p_cpu == 10 || *p_cpu == 1 || *p_cpu == 33 || *p_cpu == -5)
                                break;
                            fprintf(stderr, "p_cpu=%d  p_gpu=%d is allocated in shared "
                                   "memory.\n",*p_cpu, *p_gpu);

                            //(*fun_ptr)(1000, t, &buf[nid], &bytes_per_elem,
                            //&mem_accesses_per_elem);
                            //(*fun_ptr)(10000, t, d_buf, &bytes_per_elem,
                            //&mem_accesses_per_elem); cudaDeviceSynchronize();
                        }

                        *p_gpu = 1;
                    }

                    if ((id == 0) && (rank == 0)) {
                        startTime = getTime();
                    }

                    (*fun_ptr)(n, t, d_buf, &bytes_per_elem,
                               &mem_accesses_per_elem);

                    hipDeviceSynchronize();

                    if ((id == 0) && (rank == 0) && (t > (lim - 20))) {
                        // if ((id == 0) && (rank == 0)) {
                        // if ((id == 0) && (rank == 0) && (t > 580)) {
                        endTime = getTime();
                        seconds += (double)(endTime - startTime);
                        working_set_size = n * nthreads * nprocs;
                        total_bytes += t * working_set_size * bytes_per_elem *
                                       mem_accesses_per_elem;
                        total_flops += t * working_set_size * flop;
                        // std::cout << "nthreads: " << nthreads << " value of
                        // n: "
                        // << n << " working set size: " << working_set_size <<
                        // "total bytes: " << total_bytes << " total_flops: " <<
                        // total_flops << std::endl;
                        // total_flops += t * working_set_size * ERT_FLOP;
                    }

                    if ((id == 0) && (rank == 0) && (t == lim)) {
                        // if ((id == 0) && (rank == 0) && (t == 600)) {

                        printf("BadnWindth : %15.3lf\n", total_bytes * 1.0 /
                                                             seconds / 1024 /
                                                             1024 / 1024);
                        printf("Total Bytes: %15" PRIu64 "\n", total_bytes);
                        printf("Total flops: %15" PRIu64 "\n", total_flops);
                        printf("Total time : %15.6lf \n", seconds);

                    } // print

                    if (id == 0)
                        *p_gpu = 0;

                    hipMemcpy(&buf[nid], d_buf, n * sizeof(double),
                               hipMemcpyDeviceToHost);
                    hipDeviceSynchronize();
                }
	            if (id == 0)
                        *p_gpu = 0;


                n = nsize + 1;

                seconds = 0;
                working_set_size = 0;
                total_bytes = 0;
                total_flops = 0;

                /* while(1){
                            gpuKernel1(n/8, t, d_buf, &bytes_per_elem,
                               &mem_accesses_per_elem);
                            cudaDeviceSynchronize();
                            printf ("p=%d is allocated in shared memory.\n\n",
                   *p);


                        } */

                // if(*p_gpu == 1) break;

                /*	nNew = 1.1 * n;
            if (nNew == n) {
                    nNew = n+1;
            }

            n = nNew; */

            } // working set - nsize
        }     // argc

        hipFree(d_buf);

        if (hipGetLastError() != hipSuccess) {
            printf("Last cuda error: %s\n",
                   hipGetErrorString(hipGetLastError()));
        }

        hipDeviceReset();
    } // parallel region

    *p_gpu = 33;
    free(buf);

    return 0;
}
